#include "hip/hip_runtime.h"
//LIC// ====================================================================
//LIC// This file forms part of SciCell++, an object-oriented, 
//LIC// framework for the the simulation of biological and physical
//LIC// phenomena modelled as continuous or discrete processes.
//LIC// 
//LIC// You can find a copy at https://github.com/tachidok/scicellxx
//LIC// 
//LIC//    Version 0.6.0
//LIC//
//LIC// 31/10/2022
//LIC// 
//LIC// SciCell++ Copyright (C) 2016-2022 Julio César Pérez Sansalvador
//LIC// 
//LIC// This framework is free software; you can redistribute it and/or
//LIC// modify it under the terms of the GNU GENERAL PUBLIC LICENSE
//LIC// published by the Free Software Foundation; either version 3 of
//LIC// the License, or (at your option) any later version.
//LIC// 
//LIC// This framework is distributed in the hope that it will be useful,
//LIC// but WITHOUT ANY WARRANTY; without even the implied warranty of
//LIC// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//LIC// GNU GENERAL PUBLIC LICENSE for more details.
//LIC// 
//LIC// You should have received a copy of the GNU GENERAL PUBLIC LICENSE
//LIC// along with this framework; if not, write to the Free Software
//LIC// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA
//LIC// 02110-1301  USA.
//LIC// 
//LIC// The author may be contacted at jcp.sansalvador@inaoep.mx
//LIC// 
//LIC// ====================================================================
/// This demo uses the floor field based on the paper Kirchner, Ansgar
/// and Schadschneider, Andreas, Simulation of evacuation processes
/// using a bionics-inspired cellular automaton model for pedestrian
/// dynamics, Physica A, Elsevier, 2002.

// Include SciCell++ libraries
#include "../../../src/scicellxx.h"
#include "cuda_functions.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

// Include mTASEP algorithm
//#include "cc_mTASEP.h"

// Use the namespace of the framework
using namespace scicellxx;

// Used to define arguments
struct Args {
 argparse::ArgValue<unsigned> L;
 argparse::ArgValue<unsigned> N;
 argparse::ArgValue<Real> alpha_min;
 argparse::ArgValue<Real> alpha_max;
 argparse::ArgValue<Real> alpha_step;
 argparse::ArgValue<unsigned> alpha_n_points;
 argparse::ArgValue<Real> beta_min;
 argparse::ArgValue<Real> beta_max;
 argparse::ArgValue<Real> beta_step;
 argparse::ArgValue<unsigned> beta_n_points;
 argparse::ArgValue<Real> rho_min;
 argparse::ArgValue<Real> rho_max;
 argparse::ArgValue<Real> rho_step;
 argparse::ArgValue<unsigned> rho_n_points;
 argparse::ArgValue<Real> omega_in_min;
 argparse::ArgValue<Real> omega_in_max;
 argparse::ArgValue<Real> omega_in_step;
 argparse::ArgValue<unsigned> omega_in_n_points;
 argparse::ArgValue<Real> omega_out_min;
 argparse::ArgValue<Real> omega_out_max;
 argparse::ArgValue<Real> omega_out_step;
 argparse::ArgValue<unsigned> omega_out_n_points;
 argparse::ArgValue<unsigned> lateral_movement;
 argparse::ArgValue<unsigned> max_experiments;
 argparse::ArgValue<unsigned> max_simulations_per_experiment;
 argparse::ArgValue<unsigned> simulation_step_to_start_gathering_data;
 argparse::ArgValue<std::string> root_output_folder;
 argparse::ArgValue<unsigned> output_space_state_diagram;
 argparse::ArgValue<unsigned> output_microtubule_state;
 argparse::ArgValue<unsigned> numBlocks;
 argparse::ArgValue<unsigned> blockSize;
};

// Output parameters to a file
void output_parameters_to_file(std::string &filename, const int argc, const char **argv,  struct Args &args)
{
  // Output file
  std::ofstream output_parameters(filename, std::ios_base::out);
  
  // Write the command line as a comment into the parameters file
  output_parameters << "#";
  for (int i = 0; i < argc-1; i++)
   {
    output_parameters << argv[i] << " ";
   }
  output_parameters << argv[argc-1] << std::endl;

  const unsigned precision_real_values = 4;
  
  // output_parameters << "MPI_CORES:" << SciCellxxMPI::nprocs << std::endl;
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
    
  output_parameters << "Número de dispositivos GPU disponibles: " << deviceCount << std::endl;
  output_parameters << "L:" << args.L << std::endl;
  output_parameters << "N:" << args.N << std::endl;
  output_parameters << "alpha_min:" << setprecision(precision_real_values) << args.alpha_min << std::endl;
  output_parameters << "alpha_max:" << setprecision(precision_real_values) << args.alpha_max << std::endl;
  output_parameters << "alpha_step:" << setprecision(precision_real_values) << args.alpha_step << std::endl;
  output_parameters << "alpha_n_points:" << args.alpha_n_points << std::endl;
  output_parameters << "beta_min:" << setprecision(precision_real_values) << args.beta_min << std::endl;
  output_parameters << "beta_max:" << setprecision(precision_real_values) << args.beta_max << std::endl;
  output_parameters << "beta_step:" << setprecision(precision_real_values) << args.beta_step << std::endl;
  output_parameters << "beta_n_points:" << args.beta_n_points << std::endl;
  output_parameters << "rho_min:" << setprecision(precision_real_values) << args.rho_min << std::endl;
  output_parameters << "rho_max:" << setprecision(precision_real_values) << args.rho_max << std::endl;
  output_parameters << "rho_step:" << setprecision(precision_real_values) << args.rho_step << std::endl;
  output_parameters << "rho_n_points:" << args.rho_n_points << std::endl;
  output_parameters << "omega_in_min:" << setprecision(precision_real_values) << args.omega_in_min << std::endl;
  output_parameters << "omega_in_max:" << setprecision(precision_real_values) << args.omega_in_max << std::endl;
  output_parameters << "omega_in_step:" << setprecision(precision_real_values) << args.omega_in_step << std::endl;
  output_parameters << "omega_in_n_points:" << args.omega_in_n_points << std::endl;
  output_parameters << "omega_out_min:" << setprecision(precision_real_values) << args.omega_out_min << std::endl;
  output_parameters << "omega_out_max:" << setprecision(precision_real_values) << args.omega_out_max << std::endl;
  output_parameters << "omega_out_step:" << setprecision(precision_real_values) << args.omega_out_step << std::endl;
  output_parameters << "omega_out_n_points:" << args.omega_out_n_points << std::endl;
  output_parameters << "lateral_movement:" << args.lateral_movement << std::endl;
  output_parameters << "max_experiments:" << args.max_experiments << std::endl;
  output_parameters << "max_simulations_per_experiment:" << args.max_simulations_per_experiment << std::endl;
  output_parameters << "simulation_step_to_start_gathering_data:" << args.simulation_step_to_start_gathering_data << std::endl;
  output_parameters << "root_output_folder:" << args.root_output_folder << std::endl;
  output_parameters << "output_space_state_diagram:" << args.output_space_state_diagram << std::endl;
  output_parameters << "output_microtubule_state:" << args.output_microtubule_state << std::endl;
  output_parameters << "Número de bloques CUDA: " << args.numBlocks << std::endl;
  output_parameters << "Número de hilos por bloque: " << args.blockSize << std::endl;

  // Close the parameters file
  output_parameters.close(); 
}

/// Output boolean matrix into a csv file
void real_matrix_to_csv_file(std::vector<std::vector<Real> > m, const unsigned nrows, const unsigned ncolumns, std::string &file_name)
{
 // Create file
 std::ofstream output_file(file_name, std::ios_base::out);

 /*
 const unsigned precision_real_values = 8;
 std::ostringstream ss;
 ss << setprecision(precision_real_values);
 */
 
 for (unsigned i = 0; i < nrows; i++)
  {
   for (unsigned j = 0; j < ncolumns-1; j++)
    {
     output_file << m[i][j] << ",";
    }
   // The last element without the ','
   output_file << m[i][ncolumns-1] << std::endl;
  }
 
 // Close the file
 output_file.close();
 
} // real_matrix_to_csv_file

/// Output boolean matrix into a csv file
void boolean_matrix_to_csv_file(bool **m, const unsigned nrows, const unsigned ncolumns, std::string &file_name)
{
 // Create file
 std::ofstream output_file(file_name, std::ios_base::out);
 
 for (unsigned i = 0; i < nrows; i++)
  {
   for (unsigned j = 0; j < ncolumns-1; j++)
    {
     output_file << m[i][j] << ",";
    }
   // The last element without the ','
   output_file << m[i][ncolumns-1] << std::endl;
  }
 
 // Close the file
 output_file.close();
 
} // boolean_matrix_to_csv_file


// Definir una estructura para contener todas las estadísticas
struct Statistics {
    Real mean_density;
    Real stdev_density;
    Real median_density;
    Real mean_current;
    Real stdev_current;
    Real median_current;
};

__device__ void d_compute_mean_channels_density(const bool* m, unsigned e_m, unsigned s_m, Real* density)
{
     /*
      Computes the mean channels density of the microtubule
      (cross-sectional density/along all channels/for each column)
    */
    // Move along the microtubule and compute the mean density
    for (unsigned i = 0; i < d_L; i++)
    {
        Real sum_occupation = 0.0;

        for (unsigned k = 0; k < d_N; k++)
        {
            unsigned idx = e_m + k * d_L + i;

            sum_occupation += static_cast<Real>(m[idx]);
        }
        // Mean of sum density
        density[s_m + i] = sum_occupation / static_cast<Real>(d_N);
    }
}


// CUDA function for lateral movement
__device__ void d_try_lateral_movement(bool *m, const unsigned N, const unsigned L, const unsigned k, 
                                      const unsigned i, const unsigned n_m, const unsigned e_m)
{

    // Double-check there is a particle at the current position
    if (!m[n_m + i])
    {
        return;
    }

    // Store the indices for the microtubule above and below
    unsigned index_microtubule_above = (k == 0) ? N - 1 : k - 1;
    unsigned index_microtubule_below = (k == N - 1) ? 0 : k + 1;

    // Generate a random number to choose between the above or below microtubule
    unsigned seed = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    const Real r = hiprand_uniform(&state);

    // First choose the microtubule from above (preferred due to probability)
    if (r <= 0.5f)
    {
        // If the space at the above microtubule is free then move there!
        if (!m[e_m + index_microtubule_above * L + i])
        {
            m[n_m + i] = false;
            m[e_m + index_microtubule_above * L + i] = true;
        }
        // If the space at the below microtubule is free then move there!
        else if (!m[e_m + index_microtubule_below * L + i])
        {
            m[n_m + i] = false;
            m[e_m + index_microtubule_below * L + i] = true;
        }
    }
    // First choose the microtubule from below (preferred due to probability)
    else
    {
        if (!m[e_m + index_microtubule_below * L + i])
        {
            m[n_m + i] = false;
            m[e_m + index_microtubule_below * L + i] = true;
        }
        else if (!m[e_m + index_microtubule_above * L + i])
        {
            m[n_m + i] = false;
            m[e_m + index_microtubule_above * L + i] = true;
        }
    }
}


__device__ void d_mTASEP(bool* d_m, const unsigned e_m, const unsigned i_simulation, 
            const unsigned d_N, const unsigned d_L,
            const Real alpha, const Real beta, const Real rho,
            const Real omega_in, const Real omega_out,
            bool lateral_movement, Real &mean_current,
            Real* mean_current_per_channel,                                      
            unsigned* step_forward_particles_list,
            unsigned* step_lateral_particles_list)
{
 /*
    Applies TASEP algorithm to a multichannel microtubule

    Return the updated microtubule as an numpy matrix

    This version implements the "(d) parallel_update" strategy as
    described in section 2.1 of "The Asymmetric Exclusion Process:
    Comparison of Update Procedures, N. Rajewsky et. al., Journal of
    Statistical Physics, Vol. 92, 1998"
 */
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (i_simulation == 0){ 
      printf("\n\nEntra a d_mTASEP con el hilo %d, del experimento %d, de la simulación %d.", tid, e_m, i_simulation);
    }

    // Inicializar el estado del generador de números aleatorios
    hiprandState state;
    unsigned long long seed = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, 0, 0, &state);

    // Generar un número real aleatorio uniformemente distribuido en el rango [0,1)
    Real r = hiprand_uniform(&state);
    
    // Compute the current for each channel on the microtubule

    // Perform the method for each channel
    for (unsigned k = 0; k < d_N; k++)
    {
        unsigned n_m = e_m + k * d_L;  // Start of channel per thread 
        // *******************************************
        // Apply TASEP-LK rules
        // *******************************************
        
        // -------------------------------------------
        /// Attach (omega in)
        // -------------------------------------------
        
        // Keep track of the position of the particle attached by omega_in
        // because the just attached particle cannot move
        bool omega_in_attached_a_particle = false;
        unsigned r_pos_omega_in = 0;
        
        if (omega_in > 0.0)
        {
            // Choose a random position at the microtubule
            const unsigned r_pos = static_cast<unsigned>(hiprand_uniform(&state) * d_L);
            
            // Generate a random number
            const Real r = hiprand_uniform(&state);
            
            // if r <= omega_in and d_m[k][r_pos] == 0 then add a particle to
            // the microtubule in that position
            if (r <= omega_in && d_m[n_m + r_pos] == 0)
            {
                // Attach a particle
                d_m[n_m + r_pos] = 1;
                
                // Keep track of the position of the particle attached by omega
                // in
                r_pos_omega_in = r_pos;
                omega_in_attached_a_particle = true;
            }
        }
        
        // -------------------------------------------
        /// Detach (omega out)
        // -------------------------------------------
        
        // Keep track of the position free by omega_out becasue this
        // position cannot be occupied by other particles in the current
        // simulation step
        bool omega_out_dettached_a_particle = false;
        unsigned r_pos_omega_out = 0;
        
        if (omega_out > 0.0)
        {
            // Choose a random position at the microtubule
            const unsigned r_pos = static_cast<unsigned>(hiprand_uniform(&state) * d_L);
            
            // Generate a random number
            const Real r = hiprand_uniform(&state);
            
            // if r <= omega_out and d_m[k][r_pos] == 1 then remove the
            // particle from that position of the microtubule ALSO check that
            // the particle is not there becasue it was just attached by the
            // omega_in process
            if (r <= omega_out && d_m[n_m + r_pos] == 1 && !(omega_in_attached_a_particle && r_pos == r_pos_omega_in))
            {
            // Detach the particle
            d_m[n_m + r_pos] = 0;
            
            r_pos_omega_out = r_pos;
            omega_out_dettached_a_particle = true;
            }    
        }
        
        // *******************************************
        // Update the boundaries (left and right)
        // *******************************************
        
        // -------------------------------------------
        /// LEFT END
        // -------------------------------------------
        
        // Compute a probability to add a particle at the beginning of the
        // microtubule
        const Real a = hiprand_uniform(&state);
        // A flag indicating whether a particle was added at the beginning
        // of the microtubule
        bool added_to_start = false;
        // Is a <= alpha and the first space is free? ALSO check whether
        // the space is free due to the dettaching process by omega_out, if
        // that is the case then we cannot add a particle to the start
        if (a <= alpha && d_m[n_m] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == 0))
            {
            // Add a particle to the start
            d_m[n_m] = 1;
            // Indicate we added a particle at the beginning of the
            // microtubule so there is no need to update its position
            added_to_start = true;
            
            }
        
        // -------------------------------------------
        /// RIGHT END
        // -------------------------------------------
        
        // Compute a probability to remove the last particle of the
        // microtubule
        const Real b = hiprand_uniform(&state);
        // A flag indicating whether a particle was removed from the last
        // cell of the microtubule
        bool removed_from_end = false;
        // Is b <= beta and the last space occupied, ALSO, was the particle
        // not attached by omega_in?
        if (b <= beta && d_m[n_m + d_L-1] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == d_L-1))
        {
            // Remove the particle from the microtubule
            d_m[n_m + d_L-1] = 0;
            // Indicate we removed a particle from the last position of the
            // microtubule so other particle should not step in this cell
            removed_from_end = true;
            
        }
        
        // *******************************************
        // Update particles in the microtubule
        // *******************************************
        
        // Update particles in the microtubule using a parallel update
        // strategy (delay movement, but keep track in a list of those
        // particles to update position)
        
        // -----------------------------------------------------------
        
        // Update the first and last indexes based on the added_to_start
        // and removed_from_end flags
        unsigned end_index = d_L - 2;
        if (removed_from_end)
        {
            end_index = d_L - 3;     
        }
        
        unsigned start_index = 0;
        if (added_to_start)
        {
            start_index = 1;
        }

        unsigned forward_count = 0;
        unsigned lateral_count = 0;
        
        // // Vector with particles' indexes that will move (step-forward)
        // std::vector<unsigned> step_forward_particles_list;
        // step_forward_particles_list.reserve(d_L);
            
        // // Vector with particles' indexes that will try lateral movement
        // std::vector<unsigned> step_lateral_particles_list;
        // step_lateral_particles_list.reserve(d_L);
        
        // Update the microtubule from left to right
        for (unsigned i = start_index; i <= end_index; i++)
        {
            // Check whether there is a particle at the current cell space
            // and ensure that particle was not introduced by the omega_in
            // process
            if (d_m[n_m + i] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == i))
            {
                // Check whether there is a free space at the next cell space
                // and ensure that space is not there becase the omega_out
                // process
                if (d_m[n_m + i + 1] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == i+1))
                {
                    // Compute a probability to move to the next space
                    const Real p = hiprand_uniform(&state);
                    if (p <= rho)
                    {
                        // Add the particle index to the step forward list
                        // step_forward_particles_list.push_back(i);
                        step_forward_particles_list[n_m + forward_count] = i;
                        forward_count++;
                    }

                    // Skip the cell at the next space since there is no particle
                    // there
                    ++i;
                
                } // if (d_m[k][i+1] == 0)
                else
                {
                    // Try lateral movement since there is a particle on the next
                    // cell OR the particle on the next cell was just removed by
                    // the omega_out process
                    // step_lateral_particles_list.push_back(i);
                    step_lateral_particles_list[n_m + lateral_count] = i;
                    lateral_count++;
                    
                } // else if (d_m[k][i+1] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == i+1))
            
            } // if (d_m[k][i] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == i))

        } // for (i <= end_index)
                
        // Perform movement of particles based on the step forward list
        // Cache the size of the list
        const unsigned step_forward_particle_list_size = forward_count;

        for (unsigned j = 0; j < step_forward_particle_list_size; j++)
        {
          // Get the index of the particle
          const unsigned i = step_forward_particles_list[n_m + j];
          d_m[n_m + i] = 0;
          d_m[n_m + i + 1] = 1;

          // Add the movement to the current per channel vector
          mean_current_per_channel[n_m]+=1;
        }
        
        // *******************************************
        // Apply lateral movement
        // *******************************************
        if (d_lateral_movement)
        {
            const unsigned step_lateral_particle_list_size = lateral_count;

            for (unsigned j = 0; j < step_lateral_particle_list_size; j++)
            {
                // Get the index of the particle
                const unsigned i = step_lateral_particles_list[n_m + j];

                d_try_lateral_movement(d_m, d_N, d_L, k, i, n_m, e_m);
            }
        }
    
    } // for (k < d_N)
    
    // Compute the averaged current on all the microtubules
    mean_current = 0.0;
    for (unsigned i = 0; i < d_N; i++)
    {
        mean_current+=mean_current_per_channel[e_m + i * d_L];
    }
    
    const Real factor = 1.0/static_cast<Real>(d_L*d_N);
    
    mean_current=mean_current*factor;
}

__device__ void d_statistics_mean(Real* v, unsigned v_size, Real &mean)
{
    // Sumar todos los elementos del vector
    Real sum = 0.0;
    for (unsigned i = 0; i < v_size; ++i)
    {
        sum += v[i];
    }

    // Calcular la media
    mean = sum / static_cast<Real>(v_size);
}

__device__ void d_statistics_mean_std_median(Real* v, unsigned v_size, Real &mean, Real &stdev, Real &median)
{

    // Sumar todos los elementos del vector
    Real sum = 0.0;
    for (unsigned i = 0; i < v_size; ++i)
    {
        sum += v[ i];
    }

    // Calcular la media
    mean = sum / static_cast<Real>(v_size);

    // Calcular la desviación estándar
    Real sq_sum = 0.0;
    for (unsigned i = 0; i < v_size; ++i)
    {
        Real diff = v[ i] - mean;
        sq_sum += diff * diff;
    }
    stdev = sqrt(sq_sum / static_cast<Real>(v_size));

    // Calcular la mediana
    // Ordenar el vector (esto puede no ser eficiente)
    for (unsigned i = 0; i < v_size - 1; ++i)
    {
        for (unsigned j = 0; j < v_size - i - 1; ++j)
        {
            if (v[ j] > v[ j + 1])
            {
                Real temp = v[ j];
                v[ j] = v[ j + 1];
                v[ j + 1] = temp;
            }
        }
    }
    // Obtener la mediana
    if (v_size % 2 == 0)
    {
        median = (v[ v_size / 2 - 1] + v[ v_size / 2]) / 2.0;
    }
    else
    {
        median = v[ v_size / 2];
    }
}



// Kernel CUDA
__global__ void Run_all_configurations(Real* d_configurations, bool* d_m,  
                                        Real* d_density, Statistics* stats, 
                                        Statistics* stats_experiment,
                                        Real* mean_current_per_channel,                                      
                                        unsigned* step_forward_particles_list,
                                        unsigned* step_lateral_particles_list) {
    // Obtener el índice global del hilo
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Verificar que el hilo esté dentro del rango de configuraciones
    if (tid < d_n_all_configurations) {
        // Obtener los parámetros de configuración para este hilo
        const Real alpha = d_configurations[tid * 5 + 0];
        const Real beta = d_configurations[tid * 5 + 1];
        const Real rho = d_configurations[tid * 5 + 2];
        const Real omega_in = d_configurations[tid * 5 + 3];
        const Real omega_out = d_configurations[tid * 5 + 4];
        
        // Realizar operaciones con los parámetros de configuración
        // Por ejemplo, imprimirlos
        if (tid == 12345){ 
          printf("\nHilo %d: alpha=%f, beta=%f, rho=%f, omega_in=%f, omega_out=%f\n", tid, alpha, beta, rho, omega_in, omega_out);
        }

        unsigned experiment_counter = 0;
        
        unsigned t_m = tid * d_max_experiments * d_N * d_L;   // Start of d_m per thread

        for (unsigned i_experiment = 0; i_experiment < d_max_experiments; i_experiment++)
        {
          unsigned e_m = t_m + i_experiment * d_N * d_L;      // Start of experiment per thread 

          for (unsigned i_cell = 0; i_cell < d_N * d_L; i_cell++)
          {
            d_m[e_m + i_cell] = 0;
          }

          if (tid == 0 && i_experiment == 0){ 
            printf("\nd_m se reinició correctamente %d: ", d_m[e_m + d_N * d_L - 1]);
          }
        
          unsigned simulation_counter = 0;

          // unsigned experiment = 

          for (unsigned i_simulation_step = 0; i_simulation_step < d_max_simulations_per_experiment; i_simulation_step++)
          {
            unsigned s_m = e_m + i_simulation_step * d_L;  // Start of simulation per thread 

            // Store the current
            Real local_mean_current = 0.0;

            // Apply mTASEP
            d_mTASEP(d_m, e_m, i_simulation_step, d_N, d_L, 
                    alpha, beta, rho, omega_in, omega_out, 
                    d_lateral_movement, local_mean_current, mean_current_per_channel,                                      
                    step_forward_particles_list, step_lateral_particles_list);

            // Compute the density on the microtubule
            // d_density[e_m + i_simulation_step * d_L]
            d_compute_mean_channels_density(d_m, e_m, s_m, d_density);

            if (i_simulation_step >= d_simulation_step_to_start_gathering_data)
            {
              // Compute the mean density for this simulation step
              // (density along the microtubule)
              Real local_mean_density = 0.0;
              // SciCellxxStatistics::statistics_mean(d_density, local_mean_density);
              
              d_statistics_mean(&d_density[s_m], d_L, local_mean_density);

              unsigned tempidx = tid * d_max_experiments + i_experiment * d_tam_simulation + simulation_counter;
              // unsigned s_m = e_m + i_simulation_step * d_n_data_to_gather;

              stats_experiment[tempidx].mean_density = local_mean_density;
              stats_experiment[tempidx].mean_current = local_mean_current;

              // Increase counter
              simulation_counter++;
              
            }
          } // for (i_simulation_step < max_simulations_per_experiment)

          // Compute the mean, standard deviation and median for density on this experiment
          Real imean_density = 0.0;
          Real istdev_density = 0.0;
          Real imedian_density = 0.0;

          unsigned tempidx_exp = tid * d_max_experiments + i_experiment * d_tam_simulation;

          d_statistics_mean_std_median(&stats_experiment[tempidx_exp].mean_density, d_n_data_to_gather, imean_density, istdev_density, imedian_density);

          unsigned tempidx = tid * d_max_experiments + experiment_counter;
          
          // Keep track of the mean, standard deviation and median of the density for each experiment
          stats[tempidx].mean_density = imean_density;
          stats[tempidx].stdev_density = istdev_density;
          stats[tempidx].median_density = imedian_density;

          // Compute the mean, standard deviation and median for current on this experiment
          Real imean_current = 0.0;
          Real istdev_current = 0.0;
          Real imedian_current = 0.0;

          d_statistics_mean_std_median(&stats_experiment[tempidx_exp].mean_current, d_n_data_to_gather, imean_current, istdev_current, imedian_current);          

          stats[tempidx].mean_current = imean_current;
          stats[tempidx].stdev_current = istdev_current;
          stats[tempidx].median_current = imedian_current;        

          experiment_counter++;          

          // stats->mean_density[experiment_counter] = imean_density;
          // stats->stdev_density[experiment_counter] = istdev_density;
        } // for (i_experiment < max_experiments)

        // Compute the mean, standard deviation and median for density on this configuration
        Real imean_density = 0.0;
        Real istdev_density = 0.0;
        Real imedian_density = 0.0;

        unsigned tempidx_conf = tid * d_max_experiments;

        d_statistics_mean(&stats[tempidx_conf].mean_density, d_max_experiments,  imean_density);
        d_statistics_mean(&stats[tempidx_conf].stdev_density, d_max_experiments, istdev_density);
        d_statistics_mean(&stats[tempidx_conf].median_density, d_max_experiments, imedian_density);

        // Compute the mean, standard deviation and median for current on this configuration
        Real imean_current = 0.0;
        Real istdev_current = 0.0;
        Real imedian_current = 0.0;

        d_statistics_mean(&stats[tempidx_conf].mean_current, d_max_experiments, imean_current);
        d_statistics_mean(&stats[tempidx_conf].stdev_current, d_max_experiments, istdev_current);
        d_statistics_mean(&stats[tempidx_conf].median_current, d_max_experiments, imedian_current);

        stats[tid].mean_density = imean_density;
        stats[tid].stdev_density = istdev_density;
        stats[tid].median_density = imedian_density;
        stats[tid].mean_current = imean_current;
        stats[tid].stdev_current = istdev_current;
        stats[tid].median_current = imedian_current;
    }

    // stats[tid].mean_density = alpha + beta + rho + omega_in + omega_out;
    // stats[tid].stdev_density = alpha * beta * rho * omega_in * omega_out;
    // stats[tid].median_density = (alpha + beta + rho + omega_in + omega_out) / 5;
}

/// Computes the mean channels density (the mean of the density
/// along all channels/for each column)
std::vector<Real> compute_mean_channels_density(bool **m, const unsigned N, const unsigned L)
{
 /*
   Computes the mean channels density of the microtubule
   (cross-sectional density/along all channels/for each column)
 */

 // The vector storing the mean density (initialised with zeroes)
 std::vector<Real> density(L, 0);
 
 // Move along the microtubule and compute the mean density
 for (unsigned i = 0; i < L; i++)
  {
   Real sum_occupation = 0.0;
   for (unsigned k = 0; k < N; k++)
    {
     sum_occupation+=m[k][i];
    }
   // Mean of sum density
   density[i] = sum_occupation / Real(N);
  }

 return density;
 
}

/// Tries to perform a lateral movement on the particule at position
/// (k, i) in the microtubule
void try_lateral_movement(bool **m, const unsigned N, const unsigned L, const unsigned k, const unsigned i)
{
 /*
   Performs lateral movement on a TASEP model, this check for up/down
   possibilities, does not accounts for diagonal movement
 */
 
 // Double-check there is a particle at the current position
 if (m[k][i] == 0)
  {
   return;
  }

 // Store the indices for the microtubule above and below
 unsigned index_microtubule_above = k - 1;
 unsigned index_microtubule_below = k + 1;
 
 // Correct the indexes for the microtubules above and below
 // (periodic-boundary conditions)
 if (k == 0)
  {
   index_microtubule_above = N - 1;
  }
 
 if (k == N-1)
  {
   index_microtubule_below = 0;
  }
 
 // Used to get a seed for the random number engine
 std::random_device rd;
 // Standard mersenne_twister_engine seeded with rd()
 std::mt19937 gen(rd());
 
 // Use dist to generate a random number into a Real in the range
 // [0,1)
 std::uniform_real_distribution<> dis(0.0, 1.0);
 
 // Generate a random number to choose between the above or below
 // microtubule
 const Real r = dis(gen);
 
 // First choose the microtubule from above (preferred due to probability)
 if (r <= 0.5)
  {
   // If the space at the above microtubule is free then move there!
   if (m[index_microtubule_above][i] == false)
    {
     m[k][i] = false;
     m[index_microtubule_above][i] = true;
    }
   // If the space at the below microtubule is free then move there!
   else if (m[index_microtubule_below][i] == false)
    {
     m[k][i] = false;
     m[index_microtubule_below][i] = true;
    }
  } // if (r <= 0.5)
 else // First choose the microtubule from below (preferred due to
      // probability)
  {
   if (m[index_microtubule_below][i] == false)
    {
     m[k][i] = false;
     m[index_microtubule_below][i] = true;
    }
   else if (m[index_microtubule_above][i] == false)
    {
     m[k][i] = false;
     m[index_microtubule_above][i] = true;
    }
  } // else if (r <= 0.5)
 
}

/// Perform mTASEP method
void mTASEP(bool **m, const unsigned N, const unsigned L,
            const Real alpha, const Real beta, const Real rho,
            const Real omega_in, const Real omega_out,
            bool lateral_movement, Real &mean_current)
{
 /*
    Applies TASEP algorithm to a multichannel microtubule

    Return the updated microtubule as an numpy matrix

    This version implements the "(d) parallel_update" strategy as
    described in section 2.1 of "The Asymmetric Exclusion Process:
    Comparison of Update Procedures, N. Rajewsky et. al., Journal of
    Statistical Physics, Vol. 92, 1998"
 */
 
 // Used to get a seed for the random number engine
 std::random_device rd;
 // Standard mersenne_twister_engine seeded with rd()
 std::mt19937 gen(rd());
 
 // Use dist to generate a random number into a Real in the range
 // [0,1)
 std::uniform_real_distribution<> dis(0.0, 1.0);
 
 // Used to generate a random position in the microtubule (including
 // the first and the last position)
 std::uniform_int_distribution<> dis_microtubule_size(0, L-1);
 
 // Compute the current for each channel on the microtubule
 std::vector<unsigned> mean_current_per_channel(N, 0);
 
 // Perform the method for each channel
 for (unsigned k = 0; k < N; k++)
  {
   // *******************************************
   // Apply TASEP-LK rules
   // *******************************************
   
   // -------------------------------------------
   /// Attach (omega in)
   // -------------------------------------------
   
   // Keep track of the position of the particle attached by omega_in
   // because the just attached particle cannot move
   bool omega_in_attached_a_particle = false;
   unsigned r_pos_omega_in = 0;
   
   if (omega_in > 0.0)
    {
     // Choose a random position at the microtubule
     const unsigned r_pos = dis_microtubule_size(gen);
     
     // Generate a random number
     const Real r = dis(gen);
     
     // if r <= omega_in and m[k][r_pos] == 0 then add a particle to
     // the microtubule in that position
     if (r <= omega_in && m[k][r_pos] == 0)
      {
       // Attach a particle
       m[k][r_pos] = 1;
       
       // Keep track of the position of the particle attached by omega
       // in
       r_pos_omega_in = r_pos;
       omega_in_attached_a_particle = true;
      }
    }
   
   // -------------------------------------------
   /// Detach (omega out)
   // -------------------------------------------
   
   // Keep track of the position free by omega_out becasue this
   // position cannot be occupied by other particles in the current
   // simulation step
   bool omega_out_dettached_a_particle = false;
   unsigned r_pos_omega_out = 0;
   
   if (omega_out > 0.0)
    {
     // Choose a random position at the microtubule
     const unsigned r_pos = dis_microtubule_size(gen);
     
     // Generate a random number
     const Real r = dis(gen);
     
     // if r <= omega_out and m[k][r_pos] == 1 then remove the
     // particle from that position of the microtubule ALSO check that
     // the particle is not there becasue it was just attached by the
     // omega_in process
     if (r <= omega_out && m[k][r_pos] == 1 && !(omega_in_attached_a_particle && r_pos == r_pos_omega_in))
      {
       // Detach the particle
       m[k][r_pos] = 0;
       
       r_pos_omega_out = r_pos;
       omega_out_dettached_a_particle = true;
       
      }
     
    }
   
   // *******************************************
   // Update the boundaries (left and right)
   // *******************************************
   
   // -------------------------------------------
   /// LEFT END
   // -------------------------------------------
   
   // Compute a probability to add a particle at the beginning of the
   // microtubule
   const Real a = dis(gen);
   // A flag indicating whether a particle was added at the beginning
   // of the microtubule
   bool added_to_start = false;
   // Is a <= alpha and the first space is free? ALSO check whether
   // the space is free due to the dettaching process by omega_out, if
   // that is the case then we cannot add a particle to the start
   if (a <= alpha && m[k][0] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == 0))
    {
     // Add a particle to the start
     m[k][0] = 1;
     // Indicate we added a particle at the beginning of the
     // microtubule so there is no need to update its position
     added_to_start = true;
     
    }
   
   // -------------------------------------------
   /// RIGHT END
   // -------------------------------------------
   
   // Compute a probability to remove the last particle of the
   // microtubule
   const Real b = dis(gen);
   // A flag indicating whether a particle was removed from the last
   // cell of the microtubule
   bool removed_from_end = false;
   // Is b <= beta and the last space occupied, ALSO, was the particle
   // not attached by omega_in?
   if (b <= beta && m[k][L-1] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == L-1))
    {
     // Remove the particle from the microtubule
     m[k][L-1] = 0;
     // Indicate we removed a particle from the last position of the
     // microtubule so other particle should not step in this cell
     removed_from_end = true;
     
    }
   
   // *******************************************
   // Update particles in the microtubule
   // *******************************************
   
   // Update particles in the microtubule using a parallel update
   // strategy (delay movement, but keep track in a list of those
   // particles to update position)
   
   // -----------------------------------------------------------
   
   // Update the first and last indexes based on the added_to_start
   // and removed_from_end flags
   unsigned end_index = L - 2;
   if (removed_from_end)
    {
     end_index = L - 3;     
    }
   
   unsigned start_index = 0;
   if (added_to_start)
    {
     start_index = 1;
    }
   
   // Vector with particles' indexes that will move (step-forward)
   std::vector<unsigned> step_forward_particles_list;
   step_forward_particles_list.reserve(L);
      
   // Vector with particles' indexes that will try lateral movement
   std::vector<unsigned> step_lateral_particles_list;
   step_lateral_particles_list.reserve(L);
   
   // Update the microtubule from left to right
   for (unsigned i = start_index; i <= end_index; i++)
    {
     // Check whether there is a particle at the current cell space
     // and ensure that particle was not introduced by the omega_in
     // process
     if (m[k][i] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == i))
      {
       // Check whether there is a free space at the next cell space
       // and ensure that space is not there becase the omega_out
       // process
       if (m[k][i+1] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == i+1))
        {
         // Compute a probability to move to the next space
         const Real p = dis(gen);
         if (p <= rho)
          {
           // Add the particle index to the step forward list
           step_forward_particles_list.push_back(i);
          }

         // Skip the cell at the next space since there is no particle
         // there
         ++i;
         
        } // if (m[k][i+1] == 0)
       else
        {
         // Try lateral movement since there is a particle on the next
         // cell OR the particle on the next cell was just removed by
         // the omega_out process
         step_lateral_particles_list.push_back(i);
         
        } // else if (m[k][i+1] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == i+1))
       
      } // if (m[k][i] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == i))

    } // for (i <= end_index)
        
   // Perform movement of particles based on the step forward list
   // Cache the size of the list
   const unsigned step_forward_particle_list_size = step_forward_particles_list.size();
   for (unsigned j = 0; j < step_forward_particle_list_size; j++)
    {
     // Get the index of the particle
     const unsigned i = step_forward_particles_list[j];
     m[k][i] = 0;
     m[k][i+1] = 1;
     // Add the movement to the current per channel vector
     mean_current_per_channel[k]+=1;
    }
   
   // *******************************************
   // Apply lateral movement
   // *******************************************
   if (lateral_movement)
    {
     const unsigned step_lateral_particle_list_size = step_lateral_particles_list.size();
     for (unsigned j = 0; j < step_lateral_particle_list_size; j++)
      {
       // Get the index of the particle
       const unsigned i = step_lateral_particles_list[j];
       try_lateral_movement(m, N, L, k, i);
      }
     
    }
   
  } // for (k < N)
 
 // Compute the averaged current on all the microtubules
 mean_current = 0.0;
 for (unsigned i = 0; i < N; i++)
  {
   mean_current+=mean_current_per_channel[i];
  }
 
 const Real factor = 1.0/static_cast<Real>(L*N);
 
 mean_current=mean_current*factor;
 
}

int main(int argc, const char** argv)
{
 // Initialise scicellxx
 initialise_scicellxx();

 // Instantiate parser
 Args args;
 auto parser = argparse::ArgumentParser(argv[0], "mtasep-LK model for a multichannel microtubule with parallel update");
 
 // Add arguments
 
 // Optional
 parser.add_argument<unsigned>(args.L, "--L")
  .help("Size of the microtubule")
  .default_value("101");
 
 parser.add_argument<unsigned>(args.N, "--N")
  .help("Number of channels on the microtubule")
  .default_value("4");
 
 parser.add_argument<Real>(args.alpha_min, "--alpha_min")
  .help("The minimum probability to introduce a particle to the microtubule (at the first --left-- position of the microtubule)")
  .default_value("0.1");
 
 parser.add_argument<Real>(args.alpha_max, "--alpha_max")
  .help("The maximum probability to introduce a particle to the microtubule (at the first --left-- position of the microtubule)")
  .default_value("1.0");

 parser.add_argument<Real>(args.alpha_step, "--alpha_step")
  .help("The distance between alpha values")
  .default_value("0.1");
 
 parser.add_argument<unsigned>(args.alpha_n_points, "--alpha_n_points")
  .help("The number of points in the range alpha_min to alpha_max")
  .default_value("10");

 parser.add_argument<Real>(args.beta_min, "--beta_min")
  .help("The minimum probability to remove a particle from the microtubule (from the last --right-- position of the microtubule)")
  .default_value("0.1");

 parser.add_argument<Real>(args.beta_max, "--beta_max")
  .help("The maximum probability to remove a particle from the microtubule (from the last --right-- position of the microtubule)")
  .default_value("1.0");

 parser.add_argument<Real>(args.beta_step, "--beta_step")
  .help("The distance between beta values")
  .default_value("0.1");
 
 parser.add_argument<unsigned>(args.beta_n_points, "--beta_n_points")
  .help("The number of points in the range beta_min to beta_max")
  .default_value("10");
 
 parser.add_argument<Real>(args.rho_min, "--rho_min")
  .help("The minimum probability for a particle to step-forward")
  .default_value("0.8");
 
 parser.add_argument<Real>(args.rho_max, "--rho_max")
  .help("The maximum probability for a particle to step-forward")
  .default_value("1.0");

 parser.add_argument<Real>(args.rho_step, "--rho_step")
  .help("The distance between rho values")
  .default_value("0.1");
 
 parser.add_argument<unsigned>(args.rho_n_points, "--rho_n_points")
  .help("The number of points in the range rho_min to rho_max")
  .default_value("3");
 
 parser.add_argument<Real>(args.omega_in_min, "--omega_in_min")
  .help("The minimum probability for a particle to attach to a site in the microtubule")
  .default_value("0.0");
 
 parser.add_argument<Real>(args.omega_in_max, "--omega_in_max")
  .help("The maximum probability for a particle to attach to a site in the microtubule")
  .default_value("0.0");

 parser.add_argument<Real>(args.omega_in_step, "--omega_in_step")
  .help("The distance between omega_in values")
  .default_value("0.0");
 
 parser.add_argument<unsigned>(args.omega_in_n_points, "--omega_in_n_points")
  .help("The number of points in the range omega_in_min to omega_in_max")
  .default_value("1");
 
 parser.add_argument<Real>(args.omega_out_min, "--omega_out_min")
  .help("The minimum probability for a particle to detach from a site in the microtubule")
  .default_value("0.0");
 
 parser.add_argument<Real>(args.omega_out_max, "--omega_out_max")
  .help("The maximum probability for a particle to detach from a site in the microtubule")
  .default_value("0.0");

 parser.add_argument<Real>(args.omega_out_step, "--omega_out_step")
  .help("The distance between omega_out values")
  .default_value("0.0");
 
 parser.add_argument<unsigned>(args.omega_out_n_points, "--omega_out_n_points")
  .help("The number of points in the range omega_out_min to omega_out_max")
  .default_value("1");
 
 parser.add_argument<unsigned>(args.lateral_movement, "--lateral_movement")
  .help("Enables/disables lateral movement")
  .default_value("0");

 parser.add_argument<unsigned>(args.max_experiments, "--max_experiments")
  .help("Set the maximum number of experiments to perform")
  .default_value("1");
 
 parser.add_argument<unsigned>(args.max_simulations_per_experiment, "--max_simulations_per_experiment")
  .help("Set the maximum number of simulations per experiments")
  .default_value("200");
 
 parser.add_argument<unsigned>(args.simulation_step_to_start_gathering_data, "--simulation_step_to_start_gathering_data")
  .help("Set the simulation step to start gathering data (must be smaller than max_simulations_per_experiment)")
  .default_value("0");

 parser.add_argument<std::string>(args.root_output_folder, "--root_output_folder")
  .help("The root output folder")
  .default_value("RESLT");
 
 parser.add_argument<unsigned>(args.output_space_state_diagram, "--output_space_state_diagram")
  .help("Enables/disables output of the averaged space-time diagrams for all channels. Disable when performing large simulations")
  .default_value("0");
 
  parser.add_argument<unsigned>(args.output_microtubule_state, "--output_microtubule_state")
  .help("Enables/disables output of microtubule state (space-time diagrams for all channels). Disable when performing large simulations")
  .default_value("0");

 parser.add_argument<unsigned>(args.numBlocks, "--cudablocks")
  .help("Number of CUDA blocks")
  .default_value("32");
 
 parser.add_argument<unsigned>(args.blockSize, "--cudathreads")
  .help("Number of CUDA threads")
  .default_value("1024");
  
  // Parse the input arguments
  parser.parse_args(argc, argv);

  // Configure problem
  const unsigned L = args.L; // Size of the microtubule
  const unsigned N = args.N; // Number of channels of the microtubule
  const Real alpha_min = args.alpha_min;
  const Real alpha_max = args.alpha_max;
  const Real alpha_step = args.alpha_step;
  const unsigned alpha_n_points = args.alpha_n_points;
  const Real beta_min = args.beta_min;
  const Real beta_max = args.beta_max;
  const Real beta_step = args.beta_step;
  const unsigned beta_n_points = args.beta_n_points;
  const Real rho_min = args.rho_min;
  const Real rho_max = args.rho_max;
  const Real rho_step = args.rho_step;
  const unsigned rho_n_points = args.rho_n_points;
  const Real omega_in_min = args.omega_in_min;
  const Real omega_in_max = args.omega_in_max;
  const Real omega_in_step = args.omega_in_step;
  const unsigned omega_in_n_points = args.omega_in_n_points;
  const Real omega_out_min = args.omega_out_min;
  const Real omega_out_max = args.omega_out_max;
  const Real omega_out_step = args.omega_out_step;
  const unsigned omega_out_n_points = args.omega_out_n_points;
  bool lateral_movement = false;
  if (args.lateral_movement)
   {
    lateral_movement = true;
   }
  const unsigned max_experiments = args.max_experiments;
  const unsigned max_simulations_per_experiment = args.max_simulations_per_experiment;
  const unsigned simulation_step_to_start_gathering_data = args.simulation_step_to_start_gathering_data;
  std::string root_output_folder(args.root_output_folder);
  bool output_space_state_diagram = false;
  if (args.output_space_state_diagram)
   {
    output_space_state_diagram = true;
   }
  bool output_microtubule_state = false;
  if (args.output_microtubule_state)
   {
    output_microtubule_state = true;
   }

  // Validate parameters values
  if (simulation_step_to_start_gathering_data > max_simulations_per_experiment)
   {
    // Error message
    std::ostringstream error_message;
    error_message << "There step number to start gathering data is greater than\n"
                  << "the maximum number of simulations per experiment\n"
                  << "simulation_step_to_start_gathering_data:" << simulation_step_to_start_gathering_data
                  << "max_simulations_per_experiment:" << max_simulations_per_experiment
                  << std::endl;
    throw SciCellxxLibError(error_message.str(),
                            SCICELLXX_CURRENT_FUNCTION,
                            SCICELLXX_EXCEPTION_LOCATION);
   }

  // If the output_space_state_diagram is ENABLED check whether the
  // max_simulations_per_experiment does not exceed a MAXIMUM value
  if (output_space_state_diagram && max_simulations_per_experiment > 1000)
   {
    // Error message
    std::ostringstream error_message;
    error_message << "You enabled the output_space_state_diagram but the\n"
                  << "max_simulations_per_experiment exceeds the MAXIMUM value (1,000).\n"
                  << "This may lead to memory issues due to this large memory requirement.\n"
                  << "You can DISABLED this error check if you are sure on what you are doing!\n"
                  << std::endl;
    throw SciCellxxLibError(error_message.str(),
                            SCICELLXX_CURRENT_FUNCTION,
                            SCICELLXX_EXCEPTION_LOCATION);
   }
  
  // Output formating (files names, folders names and output to files)
  const unsigned width_number = 5;
  const char fill_char = '0';
  const unsigned precision_real_values = 4;

  // Thgis throws an error when using MPI since multile cores try to
  // create the same output folder
// #ifndef SCICELLXX_USES_MPI
  // Create output directory
  SciCellxxFileSystem::create_directory(root_output_folder);
// #endif // #ifdef SCICELLXX_USES_MPI
  
  // The string stream for the rank (used on output filenames)
  // std::ostringstream ss_rank;
  // ss_rank << SciCellxxMPI::rank;
  
  // Output parameters to a file
  // std::string parameters_filename(root_output_folder + "/parameters_r" + ss_rank.str() + ".txt");

  const unsigned numBlocks = args.numBlocks; // Number of CUDA blocks
  const unsigned blockSize = args.blockSize; // Number of CUDA threads

  std::string parameters_filename(root_output_folder + "/parameters_r" + ".txt");
  output_parameters_to_file(parameters_filename, argc, argv, args);
  
  // ------------------------------------------------------------
  // Generate all configurations as the cartesian product of the
  // ranges of parameters
  // ------------------------------------------------------------

  // Create the "set/list" with all values for each parameter

  // Alphas
  std::vector<Real> alphas;
  SciCellxxLinearSpace::create_linear_space(alphas, alpha_min, alpha_max, alpha_step, alpha_n_points);
  // Betas
  std::vector<Real> betas;
  SciCellxxLinearSpace::create_linear_space(betas, beta_min, beta_max, beta_step, beta_n_points);
  
  // Rhos
  std::vector<Real> rhos;
  SciCellxxLinearSpace::create_linear_space(rhos, rho_min, rho_max, rho_step, rho_n_points);
  
  // Omegas_in
  std::vector<Real> omegas_in;
  SciCellxxLinearSpace::create_linear_space(omegas_in, omega_in_min, omega_in_max, omega_in_step, omega_in_n_points);
  
  // Omegas_out
  std::vector<Real> omegas_out;
  SciCellxxLinearSpace::create_linear_space(omegas_out, omega_out_min, omega_out_max, omega_out_step, omega_out_n_points);
  
  // Print the information only on the master core
  // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  //  {
    scicellxx_output << "Linear spaces:" << std::endl;
    scicellxx_output << "Alphas:" << std::endl;
    SciCellxxLinearSpace::print_linear_space<Real>(alphas);
    scicellxx_output << "Betas:" << std::endl;
    SciCellxxLinearSpace::print_linear_space<Real>(betas);
    scicellxx_output << "Rhos:" << std::endl;
    SciCellxxLinearSpace::print_linear_space<Real>(rhos);
    scicellxx_output << "Omegas_in:" << std::endl;
    SciCellxxLinearSpace::print_linear_space<Real>(omegas_in);
    scicellxx_output << "Omegas_out:" << std::endl;
    SciCellxxLinearSpace::print_linear_space<Real>(omegas_out);
    scicellxx_output << std::endl;
  //  }
  
  // Create the list with the list of parameter values
  std::vector<std::vector<Real> > lists;
  lists.push_back(alphas);
  lists.push_back(betas);
  lists.push_back(rhos);
  lists.push_back(omegas_in);
  lists.push_back(omegas_out);

  // Print the information only on the master core
  // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  //  {
    scicellxx_output << "Computing cartesian product ..." << std::endl;
  //  }
  // Perform cartesian product
  std::vector<std::vector<Real> > configurations = SciCellxxCartesianProduct::product(lists);
  
  // Print the information only on the master core
  // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  //  {
    scicellxx_output << "Computing cartesian product [DONE]" << std::endl;
  //  }
  
  // Get the total number of configurations

  const unsigned n_all_configurations = configurations.size(); // Original
  // const unsigned n_all_configurations = 100;

  // Print the information only on the master core
  // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  //  {
    // Report the total number of configurations and the partitioning
    // for parallel computing
    scicellxx_output << "Total number of all configurations: " << n_all_configurations << std::endl;
    // scicellxx_output << "Number of cores: " << SciCellxxMPI::nprocs << std::endl;
  //  }

  // Print the information only on the master core
  // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  //  {
    // Only print the Cartesian product if PANIC mode is enabled
#ifdef SCICELLXX_PANIC_MODE
    // Report the cartesian product
    scicellxx_output << "Cartesian product:" << std::endl;
    SciCellxxCartesianProduct::print(configurations);
    scicellxx_output << std::endl;
#else
    scicellxx_output << "Cartesian product printing is DISABLED since PANIC_MODE is DISABLED" << std::endl;
#endif // #ifdef SCICELLXX_PANIC_MODE
  //  }
  
  // Validate that the number of cores is no larger than the number of
  // configurations
  // if ((unsigned)SciCellxxMPI::nprocs > n_all_configurations)
  //  {
  //       // Error message
  //   std::ostringstream error_message;
  //   error_message << "The number of cores is larger than the number of configurations.\n"
  //                 << "Reduce the number of cores such that each core process at least one configuration\n"
  //                 << std::endl;
  //   throw SciCellxxLibError(error_message.str(),
  //                           SCICELLXX_CURRENT_FUNCTION,
  //                           SCICELLXX_EXCEPTION_LOCATION);
  //  }
  
  // Compute the number of configurations per core
  // const unsigned n_configurations_per_core = n_all_configurations / SciCellxxMPI::nprocs;
  // const unsigned n_configurations_per_core = n_all_configurations;
  

  // The number of data to collect
  const unsigned n_data_to_gather = max_simulations_per_experiment - simulation_step_to_start_gathering_data;

  const unsigned tam_experiment = max_experiments * n_all_configurations;
  const unsigned tam_simulation = max_simulations_per_experiment * tam_experiment;
  const unsigned tam_steps = tam_experiment * N * L;

  hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(unsigned));
  hipMemcpyToSymbol(HIP_SYMBOL(d_L), &L, sizeof(unsigned));

  hipMemcpyToSymbol(HIP_SYMBOL(d_n_all_configurations), &n_all_configurations, sizeof(unsigned));
  hipMemcpyToSymbol(HIP_SYMBOL(d_max_experiments), &max_experiments, sizeof(unsigned));
  hipMemcpyToSymbol(HIP_SYMBOL(d_n_data_to_gather), &n_data_to_gather, sizeof(unsigned));

  hipMemcpyToSymbol(HIP_SYMBOL(d_max_simulations_per_experiment), &max_simulations_per_experiment, sizeof(unsigned));
  hipMemcpyToSymbol(HIP_SYMBOL(d_simulation_step_to_start_gathering_data), &simulation_step_to_start_gathering_data, sizeof(unsigned));
  hipMemcpyToSymbol(HIP_SYMBOL(d_lateral_movement), &lateral_movement, sizeof(bool));

  hipMemcpyToSymbol(HIP_SYMBOL(d_tam_experiment), &tam_experiment, sizeof(unsigned));
  hipMemcpyToSymbol(HIP_SYMBOL(d_tam_simulation), &tam_simulation, sizeof(unsigned));


  // Print the information only on the master core
  // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  //  {
  // scicellxx_output << "Number of configurations per core: " << n_configurations_per_core << std::endl;
  // //  }
  
  // // For each core get its corresponding processing configurations
  // // (the indices on the all configurations vector)
  // std::vector<unsigned> indices_configurations_per_core;
  // indices_configurations_per_core.reserve(n_configurations_per_core + 1);

  // for (unsigned i = 0; i < n_all_configurations; i++)
  //  {
  //   indices_configurations_per_core.push_back(i);
  //  } // for (i < n_all_configurations)

  // Print the indices of configurations per core
  // for (unsigned i = 0; i < indices_configurations_per_core.size(); i++)
  // {
  //  scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank,SciCellxxMPI::nprocs) << indices_configurations_per_core[i] << std::endl;
  // }
  
  // Get the real number of configuration for this core (probably
  // different from n_configurations_per_core due to rounding errors)
  // const unsigned n_configurations_this_core = indices_configurations_per_core.size();
  // scicellxx_output<< "Configuraciones por nucleo: " << n_configurations_this_core << std::endl;
  // const unsigned n_configurations_this_core = 100;
  
  
  // // Keep track of the means, standard deviation and median of the
  // // channel density space/state per configuration
  // std::vector<Real> mean_density(n_configurations_this_core);
  // std::vector<Real> stdev_density(n_configurations_this_core);
  // std::vector<Real> median_density(n_configurations_this_core);
  
  // // Keep track of the means, standard deviation and median of the
  // // microtubule current per configuration
  // std::vector<Real> mean_current(n_configurations_this_core);
  // std::vector<Real> stdev_current(n_configurations_this_core);
  // std::vector<Real> median_current(n_configurations_this_core);
  
  // unsigned config_counter = 0;

  /************************ Start CUDA ***************************/
  hipProfilerStart();

  // Create a one-dimensional array to pass to the device
  std::vector<Real> flat_configurations(n_all_configurations * 5);
  for (int i = 0; i < n_all_configurations; ++i) {
      for (int j = 0; j < 5; ++j) {
          flat_configurations[i * 5 + j] = configurations[i][j];
      }
  }

  // Crear y asignar memoria para las configuraciones en el dispositivo
  bool* d_m;
  unsigned* step_forward_particles_list;
  unsigned* step_lateral_particles_list;

  Real* d_density;
  Real* d_configurations;
  Real* mean_current_per_channel;

  Statistics* d_statistics;
  Statistics* d_statistics_experiment;

  Real* h_density = new Real[tam_simulation * L];
  Statistics* h_statistics = new Statistics[tam_experiment];
  Statistics* h_statistics_experiment = new Statistics[tam_simulation * n_data_to_gather];

  hipMalloc(&d_m, tam_experiment * N * L * sizeof(bool));
  hipMemset(d_m, 0, tam_experiment * N * L * sizeof(bool));
  hipMalloc(&d_density, tam_simulation * L * sizeof(Real));
  hipMemset(d_density, 0, tam_simulation * L * sizeof(Real));

  hipMalloc(&step_forward_particles_list, tam_steps * sizeof(unsigned));
  hipMalloc(&step_lateral_particles_list, tam_steps * sizeof(unsigned));  

  hipMalloc(&mean_current_per_channel, tam_steps * sizeof(Real));
  hipMalloc(&d_configurations, n_all_configurations * 5 * sizeof(Real));

  hipMalloc(&d_statistics, tam_experiment * sizeof(Statistics));
  hipMalloc(&d_statistics_experiment, tam_experiment * n_data_to_gather * sizeof(Statistics));

  hipMemcpy(d_configurations, flat_configurations.data(), n_all_configurations * 5 * sizeof(Real), hipMemcpyHostToDevice);

  size_t memory_max = tam_steps * sizeof(Real);
  printf("Min memory required: %.2f GB\n", 100 * (float)memory_max / (1024 * 1024 * 1024));


  scicellxx_output << "\n\t ***** Simulation starts ***** \n" << std::endl;
  
  // Llamar al kernel
  Run_all_configurations<<<numBlocks, blockSize>>>( d_configurations, 
                                                    d_m, d_density,
                                                    d_statistics,
                                                    d_statistics_experiment, 
                                                    mean_current_per_channel,
                                                    step_forward_particles_list,
                                                    step_lateral_particles_list );

  // Esperar a que todos los hilos terminen
  hipDeviceSynchronize();

  // Verificar errores
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
      printf("\n***** Error en el lanzamiento del kernel: %s\n", hipGetErrorString(error));
      // Puedes agregar más detalles de depuración aquí
      exit(-1);
  }

  scicellxx_output << "\n\n\t ***** Simulation finishes *****\n" << std::endl;

  hipMemcpy(h_density, d_density, tam_simulation * L * sizeof(Real), hipMemcpyDeviceToHost);
  hipMemcpy(h_statistics, d_statistics, n_all_configurations * sizeof(Statistics), hipMemcpyDeviceToHost);
  hipMemcpy(h_statistics_experiment, d_statistics_experiment, tam_simulation * n_data_to_gather * sizeof(Statistics), hipMemcpyDeviceToHost);

  // Imprimir los valores de h_statistics
  // for (int i = 0; i < 10; ++i) {
  //     std::cout << "Configuración " << i << ":" << std::endl;
  //     std::cout << "Mean density: " << h_statistics[i].mean_density << std::endl;
  // }

  // Liberar memoria en el dispositivo
  hipFree(step_forward_particles_list);
  hipFree(step_lateral_particles_list);
  hipFree(mean_current_per_channel);
  hipFree(d_statistics_experiment);
  hipFree(d_configurations);
  hipFree(d_statistics);
  hipFree(d_density);
  hipFree(d_m);

  /************************ CUDA end ***************************/
  hipProfilerStop();
  // // Run all configurations
  // // for (unsigned i_config = 0; i_config < n_all_configurations; i_config++)
  // for (unsigned i_config = 0; i_config < n_configurations_this_core; i_config++)
  // {
  //   const unsigned configuration_index = indices_configurations_per_core[i_config];

  //   const Real alpha = configurations[configuration_index][0];
  //   const Real beta = configurations[configuration_index][1];
  //   const Real rho = configurations[configuration_index][2];
  //   const Real omega_in = configurations[configuration_index][3];
  //   const Real omega_out = configurations[configuration_index][4];
    
  //   // Keep track of the means, standard deviation and median of the
  //   // channel density space/state per experiment
  //   std::vector<Real> mean_density_experiment(max_experiments);
  //   std::vector<Real> stdev_density_experiment(max_experiments);
  //   std::vector<Real> median_density_experiment(max_experiments);

  //   // Keep track of the means, standard deviation and median of the
  //   // microtubule current per experiment
  //   std::vector<Real> mean_current_experiment(max_experiments);
  //   std::vector<Real> stdev_current_experiment(max_experiments);
  //   std::vector<Real> median_current_experiment(max_experiments);
    
  //   unsigned experiment_counter = 0;
    
  //   // Run all experiments for the current configuration
  //   for (unsigned i_experiment = 0; i_experiment < max_experiments; i_experiment++)
  //   {
  //     // Construct the microtubule with N channels and L cells on each channel
  //     bool **m = new bool*[N];
  //     for (unsigned i_channel = 0; i_channel < N; i_channel++)
  //     {
  //       m[i_channel] = new bool[L];
  //     }

  //     // Initialize microtubule with zeroes
  //     for (unsigned i_channel = 0; i_channel < N; i_channel++)
  //     {
  //       for (unsigned i_cell = 0; i_cell < L; i_cell++)
  //       {
  //         m[i_channel][i_cell] = 0;
  //       }
  //     }
      
  //     // Keep track of the means of the mean channel density space/state
  //     std::vector<Real> mean_density_simulation(n_data_to_gather);
  //     // Keep track of the means of the current
  //     std::vector<Real> mean_current_simulation(n_data_to_gather);
  //     unsigned simulation_counter = 0;
      
  //     // Start simulation
  //     for (unsigned i_simulation_step = 0; i_simulation_step < max_simulations_per_experiment; i_simulation_step++)
  //     {
  //       // Store the current
  //       Real local_mean_current = 0.0;
  //       // Apply mTASEP
  //       mTASEP(m, N, L, alpha, beta, rho, omega_in, omega_out, lateral_movement, local_mean_current);
        
  //       // Compute the density on the microtubule
  //       std::vector<Real> mean_channels_density = compute_mean_channels_density(m, N, L);

  //       if (i_simulation_step >= simulation_step_to_start_gathering_data)
  //        {
  //         // Compute the mean density for this simulation step
  //         // (density along the microtubule)
  //         Real local_mean_density = 0.0;
  //         SciCellxxStatistics::statistics_mean(mean_channels_density, local_mean_density);
  //         // Keep track of the means for each simulation step
  //         mean_density_simulation[simulation_counter] = local_mean_density;
  //         mean_current_simulation[simulation_counter] = local_mean_current;
          
  //         // Increase counter
  //         simulation_counter++;
          
  //        }
  //     } // for (i_simulation_step < max_simulations_per_experiment)

  //     // Compute the mean, standard deviation and median for density on this experiment
  //     Real imean_density = 0.0;
  //     Real istdev_density = 0.0;
  //     Real imedian_density = 0.0;
  //     SciCellxxStatistics::statistics_mean_std_median(mean_density_simulation, imean_density, istdev_density, imedian_density);
  //     // Keep track of the mean, standard deviation and median of the density for each experiment
  //     mean_density_experiment[experiment_counter] = imean_density;
  //     stdev_density_experiment[experiment_counter] = istdev_density;
  //     median_density_experiment[experiment_counter] = imedian_density;
      
  //     // Compute the mean, standard deviation and median for current on this experiment
  //     Real imean_current = 0.0;
  //     Real istdev_current = 0.0;
  //     Real imedian_current = 0.0;
  //     SciCellxxStatistics::statistics_mean_std_median(mean_current_simulation, imean_current, istdev_current, imedian_current);
  //     // Keep track of the mean, standard deviation and median of the current for each experiment
  //     mean_current_experiment[experiment_counter] = imean_current;
  //     stdev_current_experiment[experiment_counter] = istdev_current;
  //     median_current_experiment[experiment_counter] = imedian_current;
      
  //     experiment_counter++;
      
  //     // Free memory for multichannel-microtubule
  //     for (unsigned i_channel = 0; i_channel < N; i_channel++)
  //      {
  //       delete [] m[i_channel];
  //      }
  //     delete [] m;
      
  //   } // for (i_experiment < max_experiments)
    
  //   // Compute the mean, standard deviation and median for density on this configuration
  //   Real imean_density = 0.0;
  //   Real istdev_density = 0.0;
  //   Real imedian_density = 0.0;
  //   SciCellxxStatistics::statistics_mean(mean_density_experiment, imean_density);
  //   SciCellxxStatistics::statistics_mean(stdev_density_experiment, istdev_density);
  //   SciCellxxStatistics::statistics_mean(median_density_experiment, imedian_density);
  //   // Keep track of the mean, standard deviation and median of the density for each configuration
  //   mean_density[config_counter] = imean_density;
  //   stdev_density[config_counter] = istdev_density;
  //   median_density[config_counter] = imedian_density;
    
  //   // Compute the mean, standard deviation and median for current on this configuration
  //   Real imean_current = 0.0;
  //   Real istdev_current = 0.0;
  //   Real imedian_current = 0.0;
  //   SciCellxxStatistics::statistics_mean(mean_current_experiment, imean_current);
  //   SciCellxxStatistics::statistics_mean(stdev_current_experiment, istdev_current);
  //   SciCellxxStatistics::statistics_mean(median_current_experiment, imedian_current);
  //   // Keep track of the mean, standard deviation and median of the current for each configuration
  //   mean_current[config_counter] = imean_current;
  //   stdev_current[config_counter] = istdev_current;
  //   median_current[config_counter] = imedian_current;
    
  //   config_counter++;
    
  // } // for (i_config < n_configurations_this_core)
  
  // // ****************************************************************************************
  // // Each core reports its results into a file
  // // ****************************************************************************************
  
  // // Open the file
  // // std::string output_final_results_this_core_filename(root_output_folder + "/output_r" + ss_rank.str() + ".csv");
  // // std::ofstream output_final_results_this_core_file(output_final_results_this_core_filename, std::ios_base::out);
  // // The header
  // // output_final_results_this_core_file << "id,alpha,beta,rho,omega_in,omega_out,density,std_density,median_density,current,std_current,median_current" << std::endl;
  
  // // scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "Flushing results into disk ..." << std::endl;
  // scicellxx_output << "Flushing results into disk ..." << std::endl;
  
  // // For each configuration
  // for (unsigned i_config = 0; i_config < n_configurations_this_core; i_config++)
  //  {
  //   // Get the index for the corresponding configuration for this core
  //   const unsigned configuration_index = indices_configurations_per_core[i_config];
    
  //   // Get values for each configuration
  //   const Real alpha = configurations[configuration_index][0];
  //   const Real beta = configurations[configuration_index][1];
  //   const Real rho = configurations[configuration_index][2];
  //   const Real omega_in = configurations[configuration_index][3];
  //   const Real omega_out = configurations[configuration_index][4];
     
  //   const Real imean_density = mean_density[i_config];
  //   const Real istdev_density = stdev_density[i_config];
  //   const Real imedian_density = median_density[i_config];
    
  //   const Real imean_current = mean_current[i_config];
  //   const Real istdev_current = stdev_current[i_config];
  //   const Real imedian_current = median_current[i_config];
    
  //   // Transform to string to output to file
  //   std::ostringstream ss_alpha;
  //   ss_alpha << setprecision(precision_real_values) << alpha;
  //   std::ostringstream ss_beta;
  //   ss_beta << setprecision(precision_real_values) << beta;
  //   std::ostringstream ss_rho;
  //   ss_rho << setprecision(precision_real_values) << rho;
  //   std::ostringstream ss_omega_in;
  //   ss_omega_in << setprecision(precision_real_values) << omega_in;
  //   std::ostringstream ss_omega_out;
  //   ss_omega_out << setprecision(precision_real_values) << omega_out;
    
  //   std::ostringstream ss_imean_density;
  //   ss_imean_density << setprecision(precision_real_values) << imean_density;
  //   std::ostringstream ss_istdev_density;
  //   ss_istdev_density << setprecision(precision_real_values) << istdev_density;
  //   std::ostringstream ss_imedian_density;
  //   ss_imedian_density << setprecision(precision_real_values) << imedian_density;
    
  //   std::ostringstream ss_imean_current;
  //   ss_imean_current << setprecision(precision_real_values) << imean_current;
  //   std::ostringstream ss_istdev_current;
  //   ss_istdev_current << setprecision(precision_real_values) << istdev_current;
  //   std::ostringstream ss_imedian_current;
  //   ss_imedian_current << setprecision(precision_real_values) << imedian_current;
    
  //   // output_final_results_this_core_file << configuration_index << "," << ss_alpha.str() << "," << ss_beta.str() << "," << ss_rho.str() << "," << ss_omega_in.str() << "," << ss_omega_out.str() << "," << ss_imean_density.str() << "," << ss_istdev_density.str() << "," << ss_imedian_density.str() << "," << ss_imean_current.str() << "," << ss_istdev_current.str() << "," << ss_imedian_current.str() << std::endl;
    
  //   //scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "id:" << configuration_index << "\talpha:" << ss_alpha.str() << "\tbeta:" << ss_beta.str() << "\trho:" << ss_rho.str() << "\tomega_in:" << ss_omega_in.str() << "\tomega_out:" << ss_omega_out.str() << "\tdensity:" << ss_imean_density.str() << "\tdensity(std):" << ss_istdev_density.str() << "\tdensity(median):" << ss_imedian_density.str() << "\tcurrent:" << ss_imean_current.str() << "\tcurrent(std):" << ss_istdev_current.str() << "\tcurrent(median):" << ss_imedian_current.str() << std::endl;
    
  //  } // for (i_config < n_configurations_this_core)
  
  // // Close the file
  // // output_final_results_this_core_file.close();
  
  // // scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "Flushing results into disk [DONE]" << std::endl;
  // scicellxx_output << "Flushing results into disk [DONE]" << std::endl;
  
  // // ****************************************************************************************
  // // GATHER RESULTS INTO A MASTER CORE
  // // ****************************************************************************************
  
  // // ****************************************************************************************
  // // Send the results from each core to a master core
  // // ****************************************************************************************
  
  // // Store the results into a vector to send it to a master node that
  // // will reports results in a single file
  // const unsigned n_fields_of_data_to_transfer = 12;
  // // This number incluces storage for the global id
  // Real *data_sent_to_master = new Real[n_fields_of_data_to_transfer*n_configurations_this_core];
  
  // // scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "Gathering results into a single file ..." << std::endl;
  // scicellxx_output << "Gathering results into a single file ..." << std::endl;
  
  // // For each configuration
  // for (unsigned i_config = 0; i_config < n_configurations_this_core; i_config++)
  //  {
  //   // Get the index for the corresponding configuration for this core
  //   const unsigned configuration_index = indices_configurations_per_core[i_config];
    
  //   // Get values for each configuration
  //   const Real alpha = configurations[configuration_index][0];
  //   const Real beta = configurations[configuration_index][1];
  //   const Real rho = configurations[configuration_index][2];
  //   const Real omega_in = configurations[configuration_index][3];
  //   const Real omega_out = configurations[configuration_index][4];
    
  //   const Real imean_density = mean_density[i_config];
  //   const Real istdev_density = stdev_density[i_config];
  //   const Real imedian_density = median_density[i_config];
    
  //   const Real imean_current = mean_current[i_config];
  //   const Real istdev_current = stdev_current[i_config];
  //   const Real imedian_current = median_current[i_config];

  //   const unsigned start_index = i_config*n_fields_of_data_to_transfer;
  //   data_sent_to_master[start_index + 0] = configuration_index;
  //   data_sent_to_master[start_index + 1] = alpha;
  //   data_sent_to_master[start_index + 2] = beta;
  //   data_sent_to_master[start_index + 3] = rho;
  //   data_sent_to_master[start_index + 4] = omega_in;
  //   data_sent_to_master[start_index + 5] = omega_out;
  //   data_sent_to_master[start_index + 6] = imean_density;
  //   data_sent_to_master[start_index + 7] = istdev_density;
  //   data_sent_to_master[start_index + 8] = imedian_density;
  //   data_sent_to_master[start_index + 9] = imean_current;
  //   data_sent_to_master[start_index + 10] = istdev_current;
  //   data_sent_to_master[start_index + 11] = imedian_current;
    
  //  } // for (i_config < n_configurations_this_core)
  
  // // The number of configurations to recieve from each core into master
  // int *n_configurations_to_receive_on_master_from_each_core = 0;
  
  // // On a master core gather the number of configurations on each core
  // // MPI_Gather(&n_configurations_this_core, 1, MPI_UNSIGNED,
  //           //  n_configurations_to_receive_on_master_from_each_core, 1, MPI_INT,
  //           //  SciCellxxMPI::master_core, SciCellxxMPI::comm);

  // //std::cerr << "This core configs:\n";
  // //std::cerr << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << n_configurations_this_core << std::endl;
  
  // unsigned all_configurations_mpi_reduce = 0;
  // // MPI_Reduce(&n_configurations_this_core, &all_configurations_mpi_reduce, 1, MPI_UNSIGNED, MPI_SUM,
  // //            SciCellxxMPI::master_core, SciCellxxMPI::comm);
  
  // // Validate that the sum of configurations to receive is the same as
  // // the original number of total configurations
  // // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  // //  {
  // //   if (all_configurations_mpi_reduce != n_all_configurations)
  // //    {
  // //     // Error message
  // //     std::ostringstream error_message;
  // //     error_message << "The sum of configurations to receive is different than the original\n"
  // //                   << "number of total configurations\n"
  // //                   << "(all_configurations_mpi_reduce):" << all_configurations_mpi_reduce << std::endl
  // //                   << "(n_all_configurations):" << n_all_configurations << std::endl;
  // //     throw SciCellxxLibError(error_message.str(),
  // //                             SCICELLXX_CURRENT_FUNCTION,
  // //                             SCICELLXX_EXCEPTION_LOCATION);
  // //    }
  // //  }
  
  // const unsigned n_data_sent_to_master = n_fields_of_data_to_transfer*n_configurations_this_core;
  // //std::cerr << "N data sent to master:\n";
  // //std::cerr << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << n_data_sent_to_master << std::endl;
  
  // // Vector to receive data from cores
  // Real *data_received_on_master = new Real[n_fields_of_data_to_transfer*n_all_configurations];
  
  // // // The number of data to receive on master from each core
  // // int *n_data_to_receive_on_master_from_each_core = new int[SciCellxxMPI::nprocs];
  
  // // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  // //  {
  // //   //std::cerr << "Master core configs:\n";
  // //   for (int i = 0; i < SciCellxxMPI::nprocs; i++)
  // //    {
  // //     n_data_to_receive_on_master_from_each_core[i] = n_configurations_to_receive_on_master_from_each_core[i] * n_fields_of_data_to_transfer;
  // //     //std::cerr << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs)
  // //     //          << "[" << i << "] confgs "<< n_configurations_to_receive_on_master_from_each_core[i] << std::endl;
  // //     //std::cerr << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs)
  // //     //          << "[" << i << "] data "<< n_data_to_receive_on_master_from_each_core[i] << std::endl;
  // //    }
  // //  }
  
  // // Compute the displacements vector
  // // int *n_displacement_on_mater_for_each_core = new int[SciCellxxMPI::nprocs];
  // // unsigned displ = 0;
  // // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  // //  {
  // //   for (int i = 0; i < SciCellxxMPI::nprocs; i++)
  // //    {
  // //     n_displacement_on_mater_for_each_core[i] = displ;
  // //     displ+=n_data_to_receive_on_master_from_each_core[i];
  //     //std::cerr << "N displacement on master for each core:\n";
  //     //std::cerr << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "["<<i<<"]: "<< n_displacement_on_mater_for_each_core[i] << std::endl;
  //   //  }
  // //  } // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  
  // // On a master core gather the configurations from all cores
  // // MPI_Gatherv(data_sent_to_master, n_data_sent_to_master, MPI_SC_REAL,
  // //             data_received_on_master, n_data_to_receive_on_master_from_each_core,
  // //             n_displacement_on_mater_for_each_core, MPI_SC_REAL,
  // //             SciCellxxMPI::master_core, SciCellxxMPI::comm);
  
  // // scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "Gathering results into a single file [DONE]" << std::endl;
  // scicellxx_output << "Gathering results into a single file [DONE]" << std::endl;
  
  // // ****************************************************************************************
  // // Generate a single output file with the results from all processors
  // // ****************************************************************************************  
  // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  //  {
  //   scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "Flushing gathered results into disk ..." << std::endl;
    
    // Open the file
    std::string output_final_results_filename(root_output_folder + "/output" + ".csv");
    std::ofstream output_final_results_file(output_final_results_filename, std::ios_base::out);
    // The header
    output_final_results_file << "id,alpha,beta,rho,omega_in,omega_out,density,std_density,median_density,current,std_current,median_current" << std::endl;
    
    // For each configuration
    for (unsigned i_config = 0; i_config < n_all_configurations; i_config++)
     {
      const unsigned start_index = i_config;
      
      // Get values for each configuration
      // const unsigned configuration_index = static_cast<unsigned>(data_received_on_master[start_index + 0]);
      const Real alpha = flat_configurations[start_index * 5 + 0];
      const Real beta = flat_configurations[start_index * 5 +  1];
      const Real rho = flat_configurations[start_index * 5 + 2];
      const Real omega_in = flat_configurations[start_index * 5 + 3];
      const Real omega_out = flat_configurations[start_index * 5 + 4];
      
      const Real imean_density = h_statistics[i_config].mean_density;
      const Real istdev_density = h_statistics[i_config].stdev_density;
      const Real imedian_density = h_statistics[i_config].median_density;
      
      const Real imean_current = h_statistics[i_config].mean_current;
      const Real istdev_current = h_statistics[i_config].stdev_current;
      const Real imedian_current = h_statistics[i_config].median_current;
      
      // Transform to string to output to file
      std::ostringstream ss_alpha;
      ss_alpha << setprecision(precision_real_values) << alpha;
      std::ostringstream ss_beta;
      ss_beta << setprecision(precision_real_values) << beta;
      std::ostringstream ss_rho;
      ss_rho << setprecision(precision_real_values) << rho;
      std::ostringstream ss_omega_in;
      ss_omega_in << setprecision(precision_real_values) << omega_in;
      std::ostringstream ss_omega_out;
      ss_omega_out << setprecision(precision_real_values) << omega_out;
      
      std::ostringstream ss_imean_density;
      ss_imean_density << setprecision(precision_real_values) << imean_density;
      std::ostringstream ss_istdev_density;
      ss_istdev_density << setprecision(precision_real_values) << istdev_density;
      std::ostringstream ss_imedian_density;
      ss_imedian_density << setprecision(precision_real_values) << imedian_density;
      
      std::ostringstream ss_imean_current;
      ss_imean_current << setprecision(precision_real_values) << imean_current;
      std::ostringstream ss_istdev_current;
      ss_istdev_current << setprecision(precision_real_values) << istdev_current;
      std::ostringstream ss_imedian_current;
      ss_imedian_current << setprecision(precision_real_values) << imedian_current;
      
      output_final_results_file << i_config << "," << ss_alpha.str() << "," << ss_beta.str() << "," << ss_rho.str() << "," << ss_omega_in.str() << "," << ss_omega_out.str() << "," << ss_imean_density.str() << "," << ss_istdev_density.str() << "," << ss_imedian_density.str() << "," << ss_imean_current.str() << "," << ss_istdev_current.str() << "," << ss_imedian_current.str() << std::endl;
      
      //scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "id:" << configuration_index << "\talpha:" << ss_alpha.str() << "\tbeta:" << ss_beta.str() << "\trho:" << ss_rho.str() << "\tomega_in:" << ss_omega_in.str() << "\tomega_out:" << ss_omega_out.str() << "\tdensity:" << ss_imean_density.str() << "\tdensity(std):" << ss_istdev_density.str() << "\tdensity(median):" << ss_imedian_density.str() << "\tcurrent:" << ss_imean_current.str() << "\tcurrent(std):" << ss_istdev_current.str() << "\tcurrent(median):" << ss_imedian_current.str() << std::endl;
      
     } // for (i_config < n_all_configurations)
    
    // Close the file
    output_final_results_file.close();
    
    // scicellxx_output << MPI_RANK_NPROCS_PRINT(SciCellxxMPI::rank, SciCellxxMPI::nprocs) << "Flushing gathered results into disk [DONE]" << std::endl;
    scicellxx_output<< "Flushing gathered results into disk [DONE]" << std::endl;
    
  //  } // if (SciCellxxMPI::rank == SciCellxxMPI::master_core)
  
  // Finalise chapcom
  // finalise_scicellxx();

  delete[] h_statistics;
  
  return 0;
  
}

