#include "cuda_functions.h"
#include "../../../src/scicellxx.h"
#include <hiprand/hiprand_kernel.h>


__device__ void d_compute_mean_channels_density(const bool* m, const unsigned N, const unsigned L, unsigned e_m, Real* density)
{
    // Moverse a lo largo del microtúbulo y calcular la densidad promedio
    for (unsigned i = 0; i < L; i++)
    {
        Real sum_occupation = 0.0;

        for (unsigned k = 0; k < N; k++)
        {
            // Calcular el índice del elemento actual en el arreglo plano
            unsigned idx = e_m + k * L + i;

            sum_occupation += static_cast<Real>(m[idx]);
        }
        // Calcular la densidad promedio para esta celda
        density[e_m + i * L] = sum_occupation / static_cast<Real>(N);
    }
}


// CUDA function for lateral movement
__device__ void d_try_lateral_movement(bool *m, const unsigned N, const unsigned L, const unsigned k, const unsigned i, const unsigned e_m)
{
    // Calculate the linear index
    unsigned index = e_m + k * L + i;

    // Double-check there is a particle at the current position
    if (!m[index])
    {
        return;
    }

    // Store the indices for the microtubule above and below
    unsigned index_microtubule_above = (k == 0) ? N - 1 : k - 1;
    unsigned index_microtubule_below = (k == N - 1) ? 0 : k + 1;

    // Generate a random number to choose between the above or below microtubule
    unsigned seed = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    const Real r = hiprand_uniform(&state);

    // First choose the microtubule from above (preferred due to probability)
    if (r <= 0.5f)
    {
        // If the space at the above microtubule is free then move there!
        if (!m[e_m + index_microtubule_above * L + i])
        {
            m[index] = false;
            m[e_m + index_microtubule_above * L + i] = true;
        }
        // If the space at the below microtubule is free then move there!
        else if (!m[e_m + index_microtubule_below * L + i])
        {
            m[index] = false;
            m[e_m + index_microtubule_below * L + i] = true;
        }
    }
    // First choose the microtubule from below (preferred due to probability)
    else
    {
        if (!m[e_m + index_microtubule_below * L + i])
        {
            m[index] = false;
            m[e_m + index_microtubule_below * L + i] = true;
        }
        else if (!m[e_m + index_microtubule_above * L + i])
        {
            m[index] = false;
            m[e_m + index_microtubule_above * L + i] = true;
        }
    }
}


__device__ void d_mTASEP(bool* d_m, const unsigned e_m, const unsigned i_simulation, 
            const unsigned d_N, const unsigned d_L,
            const Real alpha, const Real beta, const Real rho,
            const Real omega_in, const Real omega_out,
            bool lateral_movement, Real &mean_current,
            Real* mean_current_per_channel,                                      
            unsigned* step_forward_particles_list,
            unsigned* step_lateral_particles_list)
{
 /*
    Applies TASEP algorithm to a multichannel microtubule

    Return the updated microtubule as an numpy matrix

    This version implements the "(d) parallel_update" strategy as
    described in section 2.1 of "The Asymmetric Exclusion Process:
    Comparison of Update Procedures, N. Rajewsky et. al., Journal of
    Statistical Physics, Vol. 92, 1998"
 */
 
    // Inicializar el estado del generador de números aleatorios
    hiprandState state;
    unsigned long long seed = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, 0, 0, &state);

    // Generar un número real aleatorio uniformemente distribuido en el rango [0,1)
    Real r = hiprand_uniform(&state);

    // Generar un número entero aleatorio uniformemente distribuido en el rango [0, L-1]
    unsigned random_index = static_cast<unsigned>(hiprand_uniform(&state) * d_L);
    
    // Compute the current for each channel on the microtubule

    // Perform the method for each channel
    for (unsigned k = 0; k < d_N; k++)
    {
        // *******************************************
        // Apply TASEP-LK rules
        // *******************************************
        
        // -------------------------------------------
        /// Attach (omega in)
        // -------------------------------------------
        
        // Keep track of the position of the particle attached by omega_in
        // because the just attached particle cannot move
        bool omega_in_attached_a_particle = false;
        unsigned r_pos_omega_in = 0;
        
        if (omega_in > 0.0)
        {
            // Choose a random position at the microtubule
            const unsigned r_pos = static_cast<unsigned>(hiprand_uniform(&state) * d_L);
            
            // Generate a random number
            const Real r = hiprand_uniform(&state);
            
            // if r <= omega_in and d_m[k][r_pos] == 0 then add a particle to
            // the microtubule in that position
            if (r <= omega_in && d_m[e_m +  k * d_L + r_pos] == 0)
            {
                // Attach a particle
                d_m[e_m +  k * d_L + r_pos] = 1;
                
                // Keep track of the position of the particle attached by omega
                // in
                r_pos_omega_in = r_pos;
                omega_in_attached_a_particle = true;
            }
        }
        
        // -------------------------------------------
        /// Detach (omega out)
        // -------------------------------------------
        
        // Keep track of the position free by omega_out becasue this
        // position cannot be occupied by other particles in the current
        // simulation step
        bool omega_out_dettached_a_particle = false;
        unsigned r_pos_omega_out = 0;
        
        if (omega_out > 0.0)
        {
            // Choose a random position at the microtubule
            const unsigned r_pos = static_cast<unsigned>(hiprand_uniform(&state) * d_L);
            
            // Generate a random number
            const Real r = hiprand_uniform(&state);
            
            // if r <= omega_out and d_m[k][r_pos] == 1 then remove the
            // particle from that position of the microtubule ALSO check that
            // the particle is not there becasue it was just attached by the
            // omega_in process
            if (r <= omega_out && d_m[e_m +  k * d_L + r_pos] == 1 && !(omega_in_attached_a_particle && r_pos == r_pos_omega_in))
            {
            // Detach the particle
            d_m[e_m +  k * d_L + r_pos] = 0;
            
            r_pos_omega_out = r_pos;
            omega_out_dettached_a_particle = true;
            }    
        }
        
        // *******************************************
        // Update the boundaries (left and right)
        // *******************************************
        
        // -------------------------------------------
        /// LEFT END
        // -------------------------------------------
        
        // Compute a probability to add a particle at the beginning of the
        // microtubule
        const Real a = hiprand_uniform(&state);
        // A flag indicating whether a particle was added at the beginning
        // of the microtubule
        bool added_to_start = false;
        // Is a <= alpha and the first space is free? ALSO check whether
        // the space is free due to the dettaching process by omega_out, if
        // that is the case then we cannot add a particle to the start
        if (a <= alpha && d_m[e_m + k * d_L] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == 0))
            {
            // Add a particle to the start
            d_m[e_m + k * d_L] = 1;
            // Indicate we added a particle at the beginning of the
            // microtubule so there is no need to update its position
            added_to_start = true;
            
            }
        
        // -------------------------------------------
        /// RIGHT END
        // -------------------------------------------
        
        // Compute a probability to remove the last particle of the
        // microtubule
        const Real b = hiprand_uniform(&state);
        // A flag indicating whether a particle was removed from the last
        // cell of the microtubule
        bool removed_from_end = false;
        // Is b <= beta and the last space occupied, ALSO, was the particle
        // not attached by omega_in?
        if (b <= beta && d_m[e_m + k * d_L + d_L-1] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == d_L-1))
        {
            // Remove the particle from the microtubule
            d_m[e_m + k * d_L + d_L-1] = 0;
            // Indicate we removed a particle from the last position of the
            // microtubule so other particle should not step in this cell
            removed_from_end = true;
            
        }
        
        // *******************************************
        // Update particles in the microtubule
        // *******************************************
        
        // Update particles in the microtubule using a parallel update
        // strategy (delay movement, but keep track in a list of those
        // particles to update position)
        
        // -----------------------------------------------------------
        
        // Update the first and last indexes based on the added_to_start
        // and removed_from_end flags
        unsigned end_index = d_L - 2;
        if (removed_from_end)
        {
            end_index = d_L - 3;     
        }
        
        unsigned start_index = 0;
        if (added_to_start)
        {
            start_index = 1;
        }

        unsigned forward_count = 0;
        unsigned lateral_count = 0;
        
        // // Vector with particles' indexes that will move (step-forward)
        // std::vector<unsigned> step_forward_particles_list;
        // step_forward_particles_list.reserve(d_L);
            
        // // Vector with particles' indexes that will try lateral movement
        // std::vector<unsigned> step_lateral_particles_list;
        // step_lateral_particles_list.reserve(d_L);
        
        // Update the microtubule from left to right
        for (unsigned i = start_index; i <= end_index; i++)
        {
            // Check whether there is a particle at the current cell space
            // and ensure that particle was not introduced by the omega_in
            // process
            if (d_m[e_m + k * d_L + i] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == i))
            {
                // Check whether there is a free space at the next cell space
                // and ensure that space is not there becase the omega_out
                // process
                if (d_m[e_m + k * d_L + i + 1] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == i+1))
                {
                    // Compute a probability to move to the next space
                    const Real p = hiprand_uniform(&state);
                    if (p <= rho)
                    {
                        // Add the particle index to the step forward list
                        // step_forward_particles_list.push_back(i);
                        step_forward_particles_list[e_m + k * d_L + forward_count] = i;
                        forward_count++;
                    }

                    // Skip the cell at the next space since there is no particle
                    // there
                    ++i;
                
                } // if (d_m[k][i+1] == 0)
                else
                {
                    // Try lateral movement since there is a particle on the next
                    // cell OR the particle on the next cell was just removed by
                    // the omega_out process
                    // step_lateral_particles_list.push_back(i);
                    step_lateral_particles_list[e_m + k * d_L + lateral_count] = i;
                    lateral_count++;
                    
                } // else if (d_m[k][i+1] == 0 && !(omega_out_dettached_a_particle && r_pos_omega_out == i+1))
            
            } // if (d_m[k][i] == 1 && !(omega_in_attached_a_particle && r_pos_omega_in == i))

        } // for (i <= end_index)
                
        // Perform movement of particles based on the step forward list
        // Cache the size of the list
        const unsigned step_forward_particle_list_size = forward_count;

        for (unsigned j = 0; j < step_forward_particle_list_size; j++)
        {
        // Get the index of the particle
        const unsigned i = step_forward_particles_list[e_m + k * d_L + j];
        d_m[e_m + k * d_L + i] = 0;
        d_m[e_m + k * d_L + i + 1] = 1;
        // Add the movement to the current per channel vector
        mean_current_per_channel[e_m + k * d_L]+=1;
        }
        
        // *******************************************
        // Apply lateral movement
        // *******************************************
        if (d_lateral_movement)
        {
            const unsigned step_lateral_particle_list_size = lateral_count;

            for (unsigned j = 0; j < step_lateral_particle_list_size; j++)
            {
                // Get the index of the particle
                const unsigned i = step_lateral_particles_list[e_m + k * d_L + j];

                d_try_lateral_movement(d_m, d_N, d_L, k, i, e_m);
            }
        }
    
    } // for (k < d_N)
    
    // Compute the averaged current on all the microtubules
    mean_current = 0.0;
    for (unsigned i = 0; i < d_N; i++)
    {
        mean_current+=mean_current_per_channel[e_m + i * d_L];
    }
    
    const Real factor = 1.0/static_cast<Real>(d_L*d_N);
    
    mean_current=mean_current*factor;
}
